#include "hip/hip_runtime.h"
#include "CRTrees.hpp"
#include "util.h"
#include <experimental/random>
#include <fstream>

int
main(int argc, char** argv)
{
    if (argc != 3) {
        std::cerr << "usage: " << argv[0] << " N trails\n";
        exit(1);
    }
    const int N = atoi(argv[1]);
    const int trails = atoi(argv[2]);
    int* nn_h = new int[N];

    int *nn_d, *clusters_d, *mask_cycle_d;
    checkCudaErrors(hipMalloc(&nn_d, sizeof(int) * N));
    checkCudaErrors(hipMalloc(&clusters_d, sizeof(int) * N));
    checkCudaErrors(hipMalloc(&mask_cycle_d, sizeof(int) * N));

    CRTrees* crtrees = new CRTrees(N);

    auto c0 = std::chrono::steady_clock::now();
    auto delta_c = c0 - c0;
    for (int i=0; i<trails; ++i) {

        for (int j=0; j<N; ++j) {
            int rn = std::experimental::randint(0, N-2);
            if (rn >= j)
                rn++;
            nn_h[j] = rn;
        }
        // std::ofstream out_rand;
        // out_rand.open("rand_1nn.mtx",std::ios::trunc);
        // out_rand << "%%MatrixMarket matrix coordinate pattern general" << std::endl;
        // out_rand << N << " " << N << " "<< N * 2 << " " << std::endl;
        // for (int j=0;j<N; ++j) {
            // out_rand << nn_h[j] + 1<< " " << j + 1 << std::endl;
            // out_rand << j + 1<< " " << nn_h[j] + 1 << std::endl;
        // }
        // out_rand.close();
        checkCudaErrors(
          hipMemcpy(nn_d, nn_h, sizeof(int) * N, hipMemcpyHostToDevice));

        c0 = std::chrono::steady_clock::now();
        crtrees->get_clus(nn_d, clusters_d, N, nullptr);
        hipDeviceSynchronize();
        auto c1 = std::chrono::steady_clock::now();
        delta_c += c1 - c0;
    }
    int num_clus = crtrees->compact_clus_label(clusters_d, N);
    std::cout << "Time for CRTrees labeling to label " << N << " vertices: "
              << std::chrono::duration_cast<std::chrono::microseconds>(delta_c).count() / 1e3 / trails
              << " ms" << std::endl;
    std::cout << "number of clusters: " << num_clus << std::endl;

    int* clusters_h = new int[N];
    checkCudaErrors(hipMemcpy(
      clusters_h, clusters_d, sizeof(int) * N, hipMemcpyDeviceToHost));

    if (N <= 1024) {
        std::cout << "output labels" << std::endl;
        for (int i = 0; i < N; ++i) {
            std::cout << clusters_h[i] + 1 << " ";
        }
        std::cout << std::endl;

        int* centers = new int[N]{ 0 };
        for (int i = 0; i < N; ++i) {
            centers[clusters_h[i]] = 1;
        }

        for (int i = 0; i < N; ++i) {
            if (centers[i]) {
                std::cout << "possiable CC root: " << i + 1;
                if (i == clusters_h[i])
                    std::cout << ",  confirmed" << std::endl;
                else
                    std::cout << ",  not confirmed" << std::endl;
            }
        }
        delete[] centers;

        int* mask_cycle_h = new int[N];
        checkCudaErrors(hipMemcpy(
          mask_cycle_h, mask_cycle_d, sizeof(int) * N, hipMemcpyDeviceToHost));

        std::cout << "cycle roots" << std::endl;
        for (int i = 0; i < N; ++i)
            if (mask_cycle_h[i])
                std::cout << i + 1 << " ";
        std::cout << std::endl;
        delete[] mask_cycle_h;
    }

    checkCudaErrors(hipFree(mask_cycle_d));
    checkCudaErrors(hipFree(clusters_d));
    checkCudaErrors(hipFree(nn_d));

    delete[] clusters_h;
    delete[] nn_h;
    delete crtrees;

    return 0;
}
